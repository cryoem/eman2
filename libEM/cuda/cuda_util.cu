#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>

#include "cuda_defs.h"
#include "cuda_emfft.h"
#include "cuda_util.h"

#define MAX_THREADS 128

texture<float, 3, hipReadModeElementType> texA;
texture<float, 2, hipReadModeElementType> texA2d;
texture<float, 3, hipReadModeElementType> texB;
texture<float, 2, hipReadModeElementType> texB2d;

#include "cuda_processor.cu"
#include "cuda_cmp.cu"
#include "cuda_projector.cu"
#include "cuda_reconstructor.cu"

void cuda_bind_texture_3d(texture<float, 3, hipReadModeElementType> &texture, const hipArray * const array, const bool interp_mode) {
	texture.normalized = 0;
	if (interp_mode) texture.filterMode = hipFilterModeLinear;
	else texture.filterMode = hipFilterModePoint;
	texture.addressMode[0] = hipAddressModeClamp;
	texture.addressMode[1] = hipAddressModeClamp;
	texture.addressMode[2] = hipAddressModeClamp;
	
	hipBindTextureToArray(texture, array);
}

void cuda_bind_texture_2d(texture<float, 2, hipReadModeElementType> &texture, const hipArray * const array, const bool interp_mode) {
	texture.normalized = 0;
	if (interp_mode) texture.filterMode = hipFilterModeLinear;
	else texture.filterMode = hipFilterModePoint;
	texture.addressMode[0] = hipAddressModeClamp;
	texture.addressMode[1] = hipAddressModeClamp;

//	printf("Bound 2D texture to array %x\n", array);
	hipBindTextureToArray(texture, array);

}

hipArray* get_cuda_array(const int nx, const int ny, const int nz)
{
	hipArray *array = 0;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	
	if (nz > 1) {
		hipExtent VS = make_hipExtent(nx,ny,nz);
		hipError_t error = hipMalloc3DArray(&array, &channelDesc, VS);
		if ( error != hipSuccess) {
			printf("Could not allocate array\n");
			printf("Cuda error in allocating array: %d",int(error));
			return 0;
		}
	} else if ( ny > 1) {
		hipError_t error = hipMallocArray(&array,&channelDesc,nx,ny);
		if ( error != hipSuccess) {
			printf("Could not allocate array\n");
			printf("Cuda error in allocating array: %d",int(error));
			return 0;
		}
	}

	return array;
}

bool copy_to_array(const float * data, hipArray * array, const int nx, const int ny, const int nz, const hipMemcpyKind memkind)
{

	if (nz > 1) {
		hipExtent VS = make_hipExtent(nx,ny,nz);
		hipMemcpy3DParms copyParams = {0};
		copyParams.srcPtr   = make_hipPitchedPtr((void*)data, VS.width*sizeof(float), VS.width, VS.height);
		copyParams.dstArray = array;
		copyParams.extent   = VS;
		copyParams.kind     = memkind;
		hipError_t error =  hipMemcpy3D(&copyParams);
		if ( error != hipSuccess) {
			const char* e = hipGetErrorString(error);
			printf("CUDA error from hipMemcpy3D: %s\n",e);
			return 0;	
		}
	} else if ( ny > 1) {
		hipError_t error = hipMemcpyToArray(array, 0, 0, data, nx*ny*nz*sizeof(float), memkind);
		if ( error != hipSuccess)
		{
			const char* e = hipGetErrorString(error);
			printf("CUDA error from hipMemcpyToArray: %s\n",e);
			return 0;	
		}
	}
	return 1;
}

void bind_cuda_array_to_textureA( const hipArray* const array, const int ndims,const bool interp_mode) {
	
	if (ndims == 3) {
		cuda_bind_texture_3d(texA,array,interp_mode);
	} else {
		cuda_bind_texture_2d(texA2d,array,interp_mode);
	} 
	
}

void unbind_cuda_textureA(const int ndims) {
	if (ndims == 3) {
		hipUnbindTexture(&texA);
	}else {
		hipUnbindTexture(&texA2d);
	}
}

void bind_cuda_array_to_textureB( const hipArray* const array, const int ndims,const bool interp_mode) {
	
	if (ndims == 3) {
		cuda_bind_texture_3d(texB,array,interp_mode);
	} else {
		cuda_bind_texture_2d(texB2d,array,interp_mode);
	} 
	
}

void unbind_cuda_textureB(const int ndims) {
	if (ndims == 3) {
		hipUnbindTexture(&texB);
	}else {
		hipUnbindTexture(&texB2d);
	}
}

int getCudaDeviceManually(const int deviceCount) {
	//Set CUDA device manually if desired
	char filename[16]; // Should never be more than 12 char, but we go to 16, just to be safe. I am paranoid about buffer overflows, though in this case there isn't much risk
	if (getenv("SETCUDADEVICE") != NULL)
	{
		int i = atoi(getenv("SETCUDADEVICE"));
		if (i > deviceCount or i < 0){ printf("RUBBISH CUDA DEVICE NUMBER!!!\n"); exit(1);}
		sprintf(filename,"%s%d",cudalockfile,i); //Only works for Linux
		if (fopen(filename,"r") == NULL){
			//Put a lock on this file...
			FILE* pFile = fopen(filename,"w");
			fprintf(pFile,"%d", getpid()); // again only good for POSIX systems
			fclose(pFile);
			return i;
		} else {
			printf("DEVICE: %d already occupied\n",i);
		}
	}
	return -1;
}

int getCudaDeviceAuto(const int deviceCount) {
	//Set CUDA device automatically if desired
	//Loop through the available devices and see if any do not have a lock
	char filename[16]; // Should never be more than 12 char, but we go to 16, just to be safe. I am paranoid about buffer overflows, though in this case there isn't much risk	
	//Loop through the available devices and see if any do not have a lock
	for(int i = 0; i < deviceCount; i++)
	{
		sprintf(filename,"%s%d",cudalockfile,i); //Only works for Linux
		if (fopen(filename,"r") == NULL)
		{
			// Found a free CUDA device, now put a lock on it
			FILE* pFile = fopen(filename,"w");
			fprintf(pFile,"%d", getpid()); // again only good for POSIX systems
			fclose(pFile);
			return i;
		}
	}	
	return -1;
}

int device_init() {
	// Initialize CUDA device, if the ENV SETCUDADEVICE is set that CUDA device will be set
	// otherwise it is set based on what device is available starting with 0. If no devices are free
	// CUDA is truned off
	static bool init = true;
	int device = -1;
	
	if (init) {
		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		
		if (deviceCount == 0){
			printf("WARNING NO CUDA DEVICES FOUND, NOT USING CUDA\n");
			return device;
		}
			
 		if (deviceCount > 1) {
 			printf("%d CUDA devices detected\n",deviceCount);
 		} else { // must be one
 			printf("1 CUDA device detected\n");
 		}
		
		//try manually
		device = getCudaDeviceManually(deviceCount);
		
		//if that fails then auto
		if (device == -1){device = getCudaDeviceAuto(deviceCount);}
		
		// If no CUDA devices are free do not use CUDA
		if (device == -1)
		{
			printf("\nAll CUDA devices are occupied\nNOT using CUDA\n");
			return device;
		}
		// Otherwise set the CUDA device and check fo errors
		hipError_t cudareturn = hipSetDevice(device); 
		if(cudareturn != hipSuccess) {
			printf("\nERROR in hipSetDevice.... %s\n", hipGetErrorString(cudareturn));
			exit(2);
		} else {
			int curdev;
			hipGetDevice(&curdev);
			printf("Using CUDA device %d\n", curdev);
		}

		init = false; //Force init everytime
	}
	return device;
}

__global__ void get_edgemean_kernal(const float* data, float* edgemean, const int nx, const int ny, const int nz)
{
	int di = 0;
	float edge_sum = 0;
	float edge_mean = 0;
	size_t nxy = nx * ny;
	if (nz == 1) {
		for (int i = 0, j = (ny - 1) * nx; i < nx; ++i, ++j) {
			edge_sum += data[i] + data[j];
		}
		for (size_t i = 0, j = nx - 1; i < nxy; i += nx, j += nx) {
			edge_sum += data[i] + data[j];
		}
		edge_mean = (float)edge_sum / (nx * 2 + ny * 2);
	}
	else {
		if (nx == ny && nx == nz * 2 - 1) {
			for (size_t j = (nxy * (nz - 1)); j < nxy * nz; ++j, ++di) {
				edge_sum += data[j];
			}
		}
		else {
			for (size_t i = 0, j = (nxy * (nz - 1)); i < nxy; ++i, ++j, ++di) {
				edge_sum += data[i] + data[j];
			}
		}

		int nxy2 = nx * (ny - 1);
		for (int k = 1; k < nz - 1; ++k) {
			size_t k2 = k * nxy;
			size_t k3 = k2 + nxy2;
			for (int i = 0; i < nx; ++i, ++di) {
				edge_sum += data[i + k2] + data[i + k3];
			}
		}
		for (int k = 1; k < nz - 1; ++k) {
			size_t k2 = k * nxy;
			size_t k3 = nx - 1 + k2;
			for (int i = 1; i < ny - 1; ++i, ++di) {
				edge_sum += data[i * nx + k2] + data[i * nx + k3];
			}
		}

		edge_mean = (float)edge_sum / (di * 2);
	}
	*edgemean = edge_mean;
} 
float get_edgemean_cuda(const float* data, const int nx, const int ny, const int nz)
{

	const dim3 blockSize(1,1,1);
	const dim3 gridSize(1,1,1);

	float * d_edgemean=0;
	hipMalloc((void **)&d_edgemean, sizeof(float));
	float * h_edgemean = 0;
	h_edgemean = (float*) malloc(sizeof(float));

	get_edgemean_kernal<<<gridSize,blockSize>>>(data,d_edgemean, nx, ny, nz);
	hipDeviceSynchronize();
	hipMemcpy(h_edgemean,d_edgemean,sizeof(float),hipMemcpyDeviceToHost);
	hipFree(d_edgemean);
	
	float result = *h_edgemean;
	free(h_edgemean);
	
	return result;
}

__global__ void tovalue_kernal(float* data, const float value, const int totaltc)
{

	const uint idx = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*MAX_THREADS;

	if(idx < totaltc){
		data[idx] = value;
	}

}

void to_value_cuda(float* data, const float value, const int nx, const int ny, const int nz)
{

	int grid = int(ceil(sqrt(nx*ny*nz/MAX_THREADS)));
	
	const dim3 blockSize(MAX_THREADS,1, 1);
	const dim3 gridSize(grid,grid,1);
	tovalue_kernal<<<gridSize,blockSize>>>(data, value, nx*ny*nz);

	hipDeviceSynchronize();

	return;
}

void to_zero_cuda(float* data, const int nx, const int ny, const int nz)
{
	to_value_cuda(data, 0.0, nx, ny, nz);
}
